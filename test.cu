#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n)
  c[index] = a[index] + b[index];
}

int main(void) {
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof(int);

  // Alloc space for device
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);
  
  
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
  }